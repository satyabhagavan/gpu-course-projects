/**
 *   CS6023: GPU Programming
 *   Assignment 1
 *
 *   Please don't change any existing code in this file.
 *
 *   You can add your code whereever needed. Please add necessary memory APIs
 *   for your implementation. Use cudaFree() to free up memory as soon as you're
 *   done with an allocation. This will ensure that you don't run out of memory
 *   while running large test cases. Use the minimum required memory for your
 *   implementation. DO NOT change the kernel configuration parameters.
 */

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;

__global__ void CalculateHadamardProduct(long int *A, long int *B, int N)
{
    // TODO: Write your kernel here
    long int id = blockIdx.x * blockDim.x + threadIdx.x;
    // just want to have the index we can use this to get the
    if (id >= N * N)
    {
        // the id is out of bound we need to discard
        return;
    }

    A[(id / N) * N + id % N] = A[(id / N) * N + id % N] * B[(id % N) * N + id / N];
}
__global__ void FindWeightMatrix(long int *A, long int *B, int N)
{
    // TODO: Write your kernel here
    long int id = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    // no.of blocks before is given by blockId.x so we multiply with the dimension of each block
    if (id >= N * N)
    {
        // the Id is out of bound
        return;
    }

    if (A[(id / N) * N + id % N] > B[(id / N) * N + id % N])
    {
        // A is already larger B corresponding element
        A[(id / N) * N + id % N] = A[(id / N) * N + id % N];
    }
    else
    {
        A[(id / N) * N + id % N] = B[(id / N) * N + id % N];
    }
}

__global__ void CalculateFinalMatrix(long int *A, long int *B, int N)
{

    // TODO: Write your kernel here
    long int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    long int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    long int width = gridDim.x * blockDim.x; // total number of threads in x direction
    long int id = yIndex * width + xIndex;

    if (id >= 4 * N * N)
    {
        // the Id is out of bound
        return;
    }

    // we need to find the corresponding A matrix Id so that the multiplication can be done
    long int rowInd = id / (2 * N), colInd = id % (2 * N);
    long int a_rowInd = rowInd % N, a_colInd = colInd % N;
    B[(id / (2 * N)) * 2 * N + id % (2 * N)] = B[(id / (2 * N)) * 2 * N + id % (2 * N)] * A[a_rowInd * N + a_colInd];
}
int main(int argc, char **argv)
{

    int N;
    cin >> N;
    long int *A = new long int[N * N];
    long int *B = new long int[N * N];
    long int *C = new long int[N * N];
    long int *D = new long int[2 * N * 2 * N];

    for (long int i = 0; i < N * N; i++)
    {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++)
    {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++)
    {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++)
    {
        cin >> D[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     */

    long int *d_A;
    long int *d_B;
    long int *d_C;
    long int *d_D;

    // allocating memory
    hipMalloc(&d_A, N * N * sizeof(long int));
    hipMalloc(&d_B, N * N * sizeof(long int));
    hipMalloc(&d_C, N * N * sizeof(long int));
    hipMalloc(&d_D, 2 * N * 2 * N * sizeof(long int));

    // we need copy the matrices to the device matrices
    hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, 2 * N * 2 * N * sizeof(long int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);

    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    // we are using d_A again so we are storing product in d_A
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);

    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    // Make sure your final output from the device is stored in d_D.

    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < 2 * N; i++)
        {
            for (long int j = 0; j < 2 * N; j++)
            {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}
